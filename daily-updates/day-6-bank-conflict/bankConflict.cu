#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE_WIDTH 32
#define DATA_SIZE (TILE_WIDTH * TILE_WIDTH)

// Kernel that intentionally causes bank conflicts
__global__ void bankConflictKernel(float *input, float *output) {
    __shared__ float sharedData[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int index = ty * TILE_WIDTH + tx;

    // Access pattern causing bank conflicts
    int conflictIndex = tx * 2 % TILE_WIDTH;  // Artificial non-optimal pattern
    sharedData[ty][conflictIndex] = input[index];

    __syncthreads();

    output[index] = sharedData[ty][conflictIndex];
}

// Kernel that avoids bank conflicts by using a contiguous access pattern
__global__ void optimizedKernel(float *input, float *output) {
    __shared__ float sharedData[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int index = ty * TILE_WIDTH + tx;

    // Coalesced access pattern
    sharedData[ty][tx] = input[index];

    __syncthreads();

    output[index] = sharedData[ty][tx];
}

int main() {
    size_t size = DATA_SIZE * sizeof(float);
    float *h_input = (float*)malloc(size);
    float *h_output = (float*)malloc(size);

    // Initialize input data
    for (int i = 0; i < DATA_SIZE; i++) {
        h_input[i] = (float)(i);
    }

    float *d_input, *d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 blocksPerGrid(1, 1);

    hipEvent_t start, stop;
    float timeConflict, timeOptimized;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // --- Warm-Up for Bank Conflict Kernel ---
    bankConflictKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);
    hipDeviceSynchronize();

    // Measure bankConflictKernel execution time
    hipEventRecord(start);
    bankConflictKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeConflict, start, stop);

    // --- Warm-Up for Optimized Kernel ---
    optimizedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);
    hipDeviceSynchronize();

    // Measure optimizedKernel execution time
    hipEventRecord(start);
    optimizedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeOptimized, start, stop);

    printf("Bank Conflict Kernel Time:    %f ms\n", timeConflict);
    printf("Optimized Kernel Time:        %f ms\n", timeOptimized);

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
