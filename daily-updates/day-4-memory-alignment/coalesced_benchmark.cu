#include <hip/hip_runtime.h>
#include <stdio.h>

// Dummy kernel to flush cache
__global__ void dummyKernel(float *data, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        data[idx] = 0.0f; // Simple write to evict cache lines
    }
}

// Kernel with coalesced memory access
__global__ void coalescedKernel(const float *input, float *output, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        output[idx] = input[idx] * 2.0f;
    }
}

// Kernel with non-coalesced memory access
__global__ void nonCoalescedKernel(const float *input, float *output, int N, int stride) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int index = (idx * stride) % N; // Wrap around to match workload
        output[index] = input[index] * 2.0f;
    }
}

int main() {
    int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(float);
    hipError_t err;

    // Allocate host memory
    float *h_input = (float*)malloc(size);
    float *h_output = (float*)malloc(size);

    // Initialize host array
    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;
    }

    // Allocate device memory
    float *d_input, *d_output, *d_dummy;
    err = hipMalloc((void**)&d_input, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
    err = hipMalloc((void**)&d_output, size);
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }
    err = hipMalloc((void**)&d_dummy, size); // For cache flushing
    if (err != hipSuccess) { fprintf(stderr, "Error: %s\n", hipGetErrorString(err)); exit(EXIT_FAILURE); }

    // Configure kernel execution
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    int stride = 2;

    // Benchmark setup
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float millisecondsCoalesced = 0, millisecondsNonCoalesced = 0;

    // Benchmark coalesced kernel
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice); // Fresh input
    hipMemset(d_output, 0, size); // Fresh output
    hipEventRecord(start);
    coalescedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecondsCoalesced, start, stop);

    // Flush cache with dummy kernel
    dummyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_dummy, N);
    hipDeviceSynchronize();

    // Benchmark non-coalesced kernel
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice); // Reset input
    hipMemset(d_output, 0, size); // Reset output
    hipEventRecord(start);
    nonCoalescedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N, stride);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millisecondsNonCoalesced, start, stop);

    // Print benchmark results
    printf("Coalesced Kernel Time:    %f ms\n", millisecondsCoalesced);
    printf("Non-Coalesced Kernel Time (stride=%d): %f ms\n", stride, millisecondsNonCoalesced);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_dummy);
    free(h_input);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
