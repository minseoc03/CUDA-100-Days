#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalIdx < N) {
        C[globalIdx] = A[globalIdx] + B[globalIdx];
    }
}

int main() {
    int N = 1000000;
    size_t size = N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize host vectors
    for(int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy host vectors to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    bool success = true;
    for(int i = 0; i < N; i++) {
        if(h_C[i] != h_A[i] + h_B[i]) {
            success = false;
            printf("Error at index %d: %f != %f + %f\n", i, h_C[i], h_A[i], h_B[i]);
            break;
        }
    }

    if(success) {
        printf("Vector addition successful!\n");
    }

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
