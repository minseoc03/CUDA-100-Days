#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime>

__global__ void vectorAddGPU(float *A, float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 100000000; // 100 million
    size_t size = N * sizeof(float);
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize vectors
    for(int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipError_t err;

    err = hipMalloc(&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for A: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for B: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for C: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        exit(EXIT_FAILURE);
    }

    //intialize vector in GPU
    //if want to directly initialize vector in GPU without memcopy, need to make another kernel function.
    //using cudaMemcpy method is way more concise and easy.
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    clock_t start = clock();
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddGPU<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    //wait any process on CPU until GPU process is done.
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        exit(EXIT_FAILURE);
    }
    clock_t end = clock();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("GPU Vector Add Time: %lf seconds\n", ((double)(end - start))/CLOCKS_PER_SEC);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
